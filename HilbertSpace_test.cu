#include "hip/hip_runtime.h"
#include <catch2/catch_test_macros.hpp>
#include "HilbertSpace.hpp"
#include <random>
#include <iostream>
#include <cassert>

#define cuCHECK(call)                                                          \
	{                                                                          \
		const hipError_t error = call;                                        \
		if(error != hipSuccess) {                                             \
			printf("cuCHECK Error: %s:%d,  ", __FILE__, __LINE__);             \
			printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
			assert(error == hipSuccess);                                      \
		}                                                                      \
	};

__global__ void test_HilbertSpace_kernel(Size dim) {
	{
		// Default constructor
		{
			HilbertSpace<int> hSpace;
			assert(hSpace.dim() == 0);
		}
		HilbertSpace<int> hSpace(dim);
		assert(hSpace.dim() == dim);
	}
	{
		// Copy constructor
		HilbertSpace<int> hSpace1(dim);
		HilbertSpace<int> hSpace2(hSpace1);
		assert(hSpace1.dim() == hSpace2.dim());
	}
	{
		// Move constructor
		HilbertSpace<int> hSpace1(dim);
		HilbertSpace<int> hSpace2(std::move(hSpace1));
		assert(hSpace2.dim() == dim);
	}

	{
		// Copy assignment operator
		HilbertSpace<int> hSpace1(dim);
		HilbertSpace<int> hSpace2;
		hSpace2 = hSpace1;
		assert(hSpace1.dim() == hSpace2.dim());
	}
	{
		// Move assignment operator
		HilbertSpace<int> hSpace1(dim);
		HilbertSpace<int> hSpace2;
		hSpace2 = std::move(hSpace1);
		assert(hSpace2.dim() == dim);
	}
	{
		// Equality operator
		HilbertSpace<int> hSpace1(dim);
		HilbertSpace<int> hSpace2(hSpace1);
		assert(hSpace1.dim() == hSpace2.dim());
		assert(hSpace1 == hSpace2);
	}
}

TEST_CASE("HilbertSpace_onGPU", "test") {
	std::random_device              seed_gen;
	std::default_random_engine      engine(seed_gen());
	std::uniform_int_distribution<> dist(0, 100000);

	Size testLoop = 100;

	for(Size n = 0; n != testLoop; ++n) {
		Size dim = dist(engine);
		test_HilbertSpace_kernel<<<1, 1>>>(dim);
		cuCHECK(hipGetLastError());
		cuCHECK(hipDeviceSynchronize());
	}
}